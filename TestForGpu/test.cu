#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include<Windows.h>

using namespace std;
#pragma comment( lib,"winmm.lib" )
__global__ void Plus(float A[], float B[], float C[], int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    C[i] = A[i] + B[i];
}

//183ms
void test1()
{
    DWORD start, end;
    start = timeGetTime();

    float* A, * Ad, * B, * Bd, * C, * Cd;
    int n = 1024 * 1024;
    int size = n * sizeof(float);

    // CPU�˷����ڴ�
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // ��ʼ������
    for (int i = 0; i < n; i++)
    {
        A[i] = 90.0;
        B[i] = 10.0;
    }

    // GPU�˷����ڴ�
    hipMalloc((void**)&Ad, size);
    hipMalloc((void**)&Bd, size);
    hipMalloc((void**)&Cd, size);

    // CPU�����ݿ�����GPU��
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimBlock(512);
    dim3 dimGrid(n / 512);

    // ִ��kernel
    Plus << <dimGrid, dimBlock >> > (Ad, Bd, Cd, n);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

    // У�����
    float max_error = 0.0;
    for (int i = 0; i < n; i++)
    {
        max_error += fabs(100.0 - C[i]);
    }

    cout << "max error is " << max_error << endl;

    // �ͷ�CPU�ˡ�GPU�˵��ڴ�
    free(A);
    free(B);
    free(C);
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
    end = timeGetTime();
    cout << "total time is " << (end - start) << "ms" << endl;
   
}
__global__ void addKernel(int** C, int** A, int** B)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < 1024 && idy < 1024) {
        C[idy][idx] = A[idy][idx] + B[idy][idx];
    }
}
//14ms
void test2()
{
    DWORD start, end;
    int Row = 1024;
    int Col = 1024;
    start = timeGetTime();
    int** A = (int**)malloc(sizeof(int*) * Row);
    int** B = (int**)malloc(sizeof(int*) * Row);
    int** C = (int**)malloc(sizeof(int*) * Row);
    int* dataA = (int*)malloc(sizeof(int) * Row * Col);
    int* dataB = (int*)malloc(sizeof(int) * Row * Col);
    int* dataC = (int*)malloc(sizeof(int) * Row * Col);
    int** d_A;
    int** d_B;
    int** d_C;
    int* d_dataA;
    int* d_dataB;
    int* d_dataC;
    //malloc device memory
    hipMalloc((void**)&d_A, sizeof(int**) * Row);
    hipMalloc((void**)&d_B, sizeof(int**) * Row);
    hipMalloc((void**)&d_C, sizeof(int**) * Row);
    hipMalloc((void**)&d_dataA, sizeof(int) * Row * Col);
    hipMalloc((void**)&d_dataB, sizeof(int) * Row * Col);
    hipMalloc((void**)&d_dataC, sizeof(int) * Row * Col);
    //set value
    for (int i = 0; i < Row * Col; i++) {
        dataA[i] = 90;
        dataB[i] = 10;
    }
    //������ָ��Aָ���豸����λ�ã�Ŀ�������豸����ָ���ܹ�ָ���豸����һ��ָ��
    //A ��  dataA ���������豸�ϣ����Ƕ��߻�û�н�����Ӧ��ϵ
    for (int i = 0; i < Row; i++) {
        A[i] = d_dataA + Col * i;
        B[i] = d_dataB + Col * i;
        C[i] = d_dataC + Col * i;
    }

    hipMemcpy(d_A, A, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeof(int*) * Row, hipMemcpyHostToDevice);
    hipMemcpy(d_dataA, dataA, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB, dataB, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    dim3 threadPerBlock(16, 16);
    dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    addKernel << <blockNumber, threadPerBlock >> > (d_C, d_A, d_B);
    //������������-һ������ָ��
    hipMemcpy(dataC, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);

    int max_error = 0;
    for (int i = 0; i < Row * Col; i++)
    {
        //printf("%d\n", dataC[i]);
        max_error += abs(100 - dataC[i]);
    }

    //�ͷ��ڴ�
    free(A);
    free(B);
    free(C);
    free(dataA);
    free(dataB);
    free(dataC);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);

    printf("max_error is %d\n", max_error);
    end = timeGetTime();
    cout << "total time is " << (end - start) << "ms" << endl;

}
__global__ void matrix_mul_gpu(int* M, int* N, int* P, int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    int sum = 0;
    for (int k = 0; k < width; k++)
    {
        int a = M[j * width + k];
        int b = N[k * width + i];
        sum += a * b;
    }
    P[j * width + i] = sum;
}
//234ms
void test3()
{
    DWORD start, end;
    int Row = 1024;
    int Col = 1024;
    start = timeGetTime();

    int* A = (int*)malloc(sizeof(int) * Row * Col);
    int* B = (int*)malloc(sizeof(int) * Row * Col);
    int* C = (int*)malloc(sizeof(int) * Row * Col);
    //malloc device memory
    int* d_dataA, * d_dataB, * d_dataC;
    hipMalloc((void**)&d_dataA, sizeof(int) * Row * Col);
    hipMalloc((void**)&d_dataB, sizeof(int) * Row * Col);
    hipMalloc((void**)&d_dataC, sizeof(int) * Row * Col);
    //set value
    for (int i = 0; i < Row * Col; i++) {
        A[i] = 90;
        B[i] = 10;
    }

    hipMemcpy(d_dataA, A, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB, B, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    dim3 threadPerBlock(16, 16);
    dim3 blockNumber((Col + threadPerBlock.x - 1) / threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y);
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA, d_dataB, d_dataC, Col);
    //������������-һ������ָ��
    hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);

    //�ͷ��ڴ�
    free(A);
    free(B);
    free(C);
    hipFree(d_dataA);
    hipFree(d_dataB);
    hipFree(d_dataC);

    end = timeGetTime();
    cout << "total time is " << (end - start) << "ms" << endl;

}

int main()
{
    test1();
    test2();
    test3();
    return 0;
}